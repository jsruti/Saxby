#include "hip/hip_runtime.h"
/*
* Compile: nvcc -o saxby saxby.cu
* Run: ./saxby
*/
#include <stdio.h>
__global__ void
daxbyAdd(const float *A, const float *B, float *C, float x,int numElements){
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i < numElements){
		C[i] = A[i]* x + B[i];
	}
}
int main (void){
	int N = 1<<20;
	float *x, *y, *z, *d_x, *d_y, *d_z;
	x = (float*) malloc(N*sizeof(float));
	y = (float*) malloc(N*sizeof(float));
	z = (float*) malloc(N*sizeof(float));

	hipMalloc(&d_x, N*sizeof(float));
	hipMalloc(&d_y, N*sizeof(float));
	hipMalloc(&d_z, N*sizeof(float));
	
	for(int i = 0; i < N; i++){
		x[i] = 1.0f;
		y[i] = 1.0f;
		//z[i] = 2.0f;
	}

	hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);
	//hipMemcpy(d_z, z, N*sizeof(float), hipMemcpyHostToDevice);

	daxbyAdd<<<(N+255)/256, 256>>>(d_x, d_y, d_z, 2.0f, N);

	hipMemcpy(z, d_z, N*sizeof(float), hipMemcpyDeviceToHost);

	float maxError = 0.0f;
	for(int i = 0; i < N; i++) {
		maxError = max(maxError, abs(y[i] - 3.0f));
	}
	printf("Max Error: %f\n", maxError);

	hipFree(d_x);
	hipFree(d_y);
	hipFree(d_z);
	free(x);
	free(y);
	free(z);
}
